#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "cubeai/base/cubeai_config.h"

#ifdef USE_CUDA

#include "cubeai/base/cubeai_types.h"

#include <boost/log/trivial.hpp>
#include <iostream>

namespace exe{
	
	using cubeai::float_t;

__global__ void sum(float_t* v1, float_t* v2, float_t* v3){

	// the thread id we use to correctly
	// access the vector prosition we
	// are interested in
	int idx = threadIdx.x;
	float_t f1 = v1[idx];
	float_t f2 = v2[idx];
	float_t f3 = f1 + f2;
	v3[idx] = f3;

}

}

int main() {
	
	using namespace exe;
	
	BOOST_LOG_TRIVIAL(info)<<"Running example...";
    
	const int ARRAY_SIZE = 64;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float_t);

	// host arrays
	float_t h_v1[ARRAY_SIZE];
	float_t h_v2[ARRAY_SIZE];
	float_t h_v3[ARRAY_SIZE];
	
	for(int i=0; i<ARRAY_SIZE; ++i){
	   h_v1[i] = float_t(i);
	   h_v2[i] = float_t(i);
	   h_v3[i] = 0.0f;
	}

	// device arrays
	float_t* d_v1 = nullptr;
	float_t* d_v2 = nullptr;
	float_t* d_v3 = nullptr;

	// allocate GPU memory for the device arrays
	hipMalloc((void **) &d_v1, ARRAY_BYTES);
	hipMalloc((void **) &d_v2, ARRAY_BYTES);
	hipMalloc((void **) &d_v3, ARRAY_BYTES);

	// transfer array to GPU 
	hipMemcpy(d_v1, h_v1, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_v2, h_v2, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_v3, h_v3, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch kernel
	sum<<<1, ARRAY_SIZE>>>(d_v1, d_v2, d_v3);

	// copy the output from the GPU to host
	hipMemcpy(h_v3, d_v3, ARRAY_BYTES, hipMemcpyDeviceToHost);


	for(int i=0; i<ARRAY_SIZE; ++i){
		std::cout<<h_v1[i]<<"+"<<h_v2[i]<<"="<<h_v3[i]<<std::endl;
	}

	// free memory
	hipFree(d_v1);
	hipFree(d_v2);
	hipFree(d_v3);
		
    hipDeviceReset();
	
	BOOST_LOG_TRIVIAL(info)<<"Done...";
    return 0;
}

#else
#include <iostream>
int main() {
	std::cout<<"This example requires CUDA support enabled. Reconfigure CubeRL and set USE_CUDA=ON"<<std::endl;
	return 1;
}
#endif